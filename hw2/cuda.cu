// cuda.cu
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

__global__ void vector_multiply_row_device(float * a, float * b, float * c, int m)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int j;
    for (j=0; j<m; j++)
    {
        a[idx] += b[j+idx*m]*c[j];
    }
}
int main(void)
{
    float *a_h, *b_h, *c_h;            // pointers to host memory
    float *a_d, *b_d, *c_d;            // pointers to device memory
    int i, j, m, n;

    printf("Please give m and n: ");
    scanf("%d %d",&m,&n);

    struct timeval start, end;
    gettimeofday(&start, NULL);

    // allocate arrays on host
    if ( (a_h=(float *)malloc(m*sizeof(float))) == NULL )
        perror("memory allocation for a");
    if ( (b_h=(float *)malloc(m*n*sizeof(float))) == NULL )
        perror("memory allocation for b");
    if ( (c_h=(float *)malloc(n*sizeof(float))) == NULL )
        perror("memory allocation for c");

    // allocate array on device 
    hipMalloc((void **) &a_d, n*sizeof(float));
    hipMalloc((void **) &b_d, m*n*sizeof(float));
    hipMalloc((void **) &c_d, n*sizeof(float));

    // initialization of host data
    printf("Initializing matrix B and vector c\n");
    for (j=0; j<n; j++)
        c_h[j] = 2.0;
    for (i=0; i<m; i++)
        for (j=0; j<n; j++)
            b_h[i*n+j] = i;

    printf("Vector c:\n");
    for (j=0; j<n; j++)
        printf("c_h[%d] = %f\n", j, c_h[j]);

    printf("Matrix B:\n");
    for (i=0; i<m; i++)
        for (j=0; j<n; j++)
            printf("b_h[%d] = %f\n", i*n+j, b_h[i*n+j]);

    printf("Initializing a to 0\n");
    for(i=0; i<n; i++)
        a_h[i] = 0.0;

    // copy data from host to device
    hipMemcpy(a_d, a_h, m*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, n*m*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_d, c_h, m*sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 4;
    int numBlocks = m;

    vector_multiply_row_device <<< blockSize, numBlocks >>> (a_d, b_d, c_d, m);

    hipMemcpy(a_h, a_d, m*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b_h, b_d, n*m*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c_h, c_d, m*sizeof(float), hipMemcpyDeviceToHost);

    gettimeofday(&end, NULL);
    printf("Elapsed time: %ldus\n", ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec)));

    printf("Vector a_h:\n");
    for (j=0; j<n; j++)
        printf("a_h[%d] = %f\n", j, a_h[j]);

    // cleanup
    free(a_h); free(b_h); free(c_h); hipFree(a_d); hipFree(b_d); hipFree(c_d); 
}
