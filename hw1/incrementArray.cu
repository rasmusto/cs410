// incrementArray.cu
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
void incrementArrayOnHost(float *a, int N)
{
    int i;
    for (i=0; i < N; i++) a[i] = a[i]+1.f;
}
__global__ void incrementArrayOnDevice(float *a, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<N) a[idx] = a[idx]+1.f;
}
int main(void)
{
    float *a_h, *b_h;           // pointers to host memory
    float *a_d;                 // pointer to device memory
    int i, N = 20;
    size_t size = N*sizeof(float);
    // allocate arrays on host
    a_h = (float *)malloc(size);
    b_h = (float *)malloc(size);
    // allocate array on device 
    hipMalloc((void **) &a_d, size);
    // initialization of host data
    for (i=0; i<N; i++) a_h[i] = (float)i;
    // copy data from host to device
    hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);

    printf("Original array:\n");
    for(i=0; i<N; i++)
        printf("a_h[%d]\t= %f\n", i, a_h[i]);
    printf("\n");

    // do calculation on host
    incrementArrayOnHost(a_h, N);

    printf("Host array (incremented):\n");
    for(i=0; i<N; i++)
        printf("a_h[%d]\t= %f\n", i, a_h[i]);
    printf("\n");

    // do calculation on device:
    // Part 1 of 2. Compute execution configuration
    int blockSize = 4;
    int nBlocks = N/blockSize + (N%blockSize == 0?0:1);

    // Part 2 of 2. Call incrementArrayOnDevice kernel 
    incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
    // Retrieve result from device and store in b_h
    hipMemcpy(b_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    // check results
    for (i=0; i<N; i++) assert(a_h[i] == b_h[i]);

    printf("Device array (incremented):\n");
    for(i=0; i<N; i++)
        printf("b_h[%d]\t= %f\n", i, b_h[i]);
    printf("\n");

    // cleanup
    free(a_h); free(b_h); hipFree(a_d); 
}
